#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>
#define Num 500000
#define PrimeNumMax 5000000
#define GroupSize 8

__inline__ __device__ int isPrime(unsigned int* prime, unsigned long long a)
{
	int c0(0);
	for (;;)
	{
		unsigned int p(prime[c0++]);
		if (unsigned long long(p) * p > a)return 1;
		if (a % p == 0)return 0;
	}
}
__inline__ __device__ unsigned long long qml(unsigned long long a, unsigned long long b, unsigned long long m)
{
	unsigned long long s = 0;
	while (b)
	{
		if (b & 1) s = (s + a) % m;
		a = (a + a) % m;
		b >>= 1;
	}
	return s;
}
__inline__ __device__ unsigned long long mypow(unsigned long long a, unsigned long long b, unsigned long long m)
{
	unsigned long long s = 1;
	while (b)
	{
		if (b & 1) s = qml(s, a, m);
		a = qml(a, a, m);
		b >>= 1;
	}
	return s;
}
__inline__ __device__ int Miller_Rabbin(unsigned long long x, hiprandState* state)
{
	if (x == 2) return true;
	for (int i = 0; i < 2; ++i)
	{
		unsigned long long a = hiprand(state) % (x - 2) + 2;
		if (mypow(a, x - 1, x) != 1)
			return 0;
	}
	return 1;
}
__inline__ __device__ int Miller_Rabbin_Op(unsigned long long x)
{
	if (x == 2) return true;
	unsigned int table[7]{ 2,3,5,7,11,13,17 };
	for (int c0(0); c0 < 7; ++c0)
	{
		unsigned long long a = table[c0];
		if (mypow(a, x - 1, x) != 1)
			return 0;
	}
	return 1;
}


__global__ void initRandom(hiprandState* state, unsigned int seed)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, state + id);
}

__global__ void fuckCPU(unsigned int* prime, unsigned int* answer)
{
	unsigned int id(threadIdx.x + blockIdx.x * blockDim.x);
	unsigned int upper((id + 1) * GroupSize + 2);
	unsigned int limit((upper > Num + 1) ? (Num + 1) : upper);
	unsigned int num(0);
	for (unsigned int c0(id* GroupSize + 2); c0 < limit; ++c0)
	{
		unsigned long long a(c0);
		a = 2 * a * a - 1;
		num += isPrime(prime, a);
	}
	answer[id] = num;
}
__global__ void fuckCPU_Op1(unsigned int* prime, unsigned int* answer)//blocksize: 64
{
	__shared__ unsigned int primeS[8192];
	for (int c0(0); c0 < 64; ++c0)
	{
		unsigned int id(threadIdx.x + c0 * 128);
		primeS[id] = prime[id];
		primeS[id + 64] = prime[id + 64];
	}
	__syncthreads();
	unsigned int id(threadIdx.x + blockIdx.x * blockDim.x);
	unsigned int upper((id + 1) * GroupSize + 2);
	unsigned int limit((upper > Num + 1) ? (Num + 1) : upper);
	unsigned int num(0);
	for (unsigned int c0(id* GroupSize + 2); c0 < limit; ++c0)
	{
		unsigned long long a(c0);
		a = 2 * a * a - 1;
		int c1(0);
		for (;;)
		{
			unsigned int p;
			if (c1 < 8192)p = primeS[c1++];
			else p = prime[c1++];
			if (unsigned long long(p) * p > a)
			{
				num += 1;
				break;
			}
			if (a % p == 0)break;
		}
	}
	answer[id] = num;
}
__global__ void fuckCPU_Op2(unsigned int* answer)//, hiprandState* state)
{
	unsigned int table[24] = { 2,3,5,7,11,13,17,19,23,29,31,
		37,41,43,47,53,59,61,67,71,73,79,83,89 //,97,101,103,
		//107,109,113,127,131,137,139,149,151,157,163,167,173,
		//179,181,191,193,197,199,211,223,227,229
	};
	unsigned int id(threadIdx.x + blockIdx.x * blockDim.x);
	unsigned int upper((id + 1) * GroupSize + 2);
	unsigned int limit((upper > Num + 1) ? (Num + 1) : upper);
	unsigned int num(0);
	//state += id;
	for (unsigned int c0(id* GroupSize + 2); c0 < limit; ++c0)
	{
		unsigned long long a(c0);
		a = 2 * a - 1;
		int c1(0);
		for (; c1 < 24; ++c1)
			if (a % table[c1] == 0)
			{
				if (a == table[c1])num++;
				break;
			}
		if (c1 == 24)
			num += Miller_Rabbin_Op(a);
	}
	answer[id] = num;
}

unsigned int call(unsigned int* prime)
{
	size_t answerSize(68 * 1024 * 4);
	//size_t stateSize(272 * 1024 * sizeof(hiprandState));
	//unsigned int* primeGPU;
	unsigned int* answerGPU;
	unsigned int* answerCPU((unsigned int*)::malloc(answerSize));
	//hiprandState* state;
	//hipMalloc(&primeGPU, PrimeNumMax * 4);
	hipMalloc(&answerGPU, answerSize);
	//hipMalloc(&state, stateSize);
	//hipMemcpy(primeGPU, prime, PrimeNumMax * 4, hipMemcpyHostToDevice);
	//fuckCPU_Op1 << <68, 64 >> > (primeGPU, answerGPU);
	//initRandom << <272, 1024 >> > (state, rand());
	fuckCPU_Op2 << <68, 1024 >> > (answerGPU);
	//fuckCPU_Op2 << <272, 1024 >> > (answerGPU, state);
	hipMemcpy(answerCPU, answerGPU, answerSize, hipMemcpyDeviceToHost);
	unsigned int answer(0);
	for (int c0(0); c0 < answerSize / 4; ++c0)
		answer += answerCPU[c0];
	//hipFree(primeGPU);
	hipFree(answerGPU);
	//hipFree(state);
	free(answerCPU);
	return answer;
}